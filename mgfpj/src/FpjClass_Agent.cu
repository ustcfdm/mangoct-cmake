#include "hip/hip_runtime.h"
#include "FpjClass_Agent.cuh"
#include <stdio.h>
#include "stdafx.h"

#define PI 3.1415926536f
#define STEPSIZE 0.2f

__global__ void InitDistance(float *distance_array, const float distance, const int V)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < V)
	{
		distance_array[tid] = distance;
	}
}

__global__ void InitU(float* u, const int N, const float du, const float offcenter)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < N)
	{
		u[tid] = (tid - (N - 1) / 2.0f) * du + offcenter;
	}
}

__global__ void InitBeta(float* beta, const int V, const float startAngle, const float totalScanAngle)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid<V)
	{
		beta[tid] = (totalScanAngle / V * tid + startAngle) * PI / 180.0f;
	}
}

// img: image data
// sgm: sinogram data
// u: array of each detector element position
// beta: array of each view angle [radian]
// M: image dimension
// S: number of image slices
// N_z: number of detector elements in Z direction
// N: number of detector elements (sinogram width)
// V: number of views (sinogram height)
// dx: image pixel size [mm]
// dz: image slice thickness [mm]
// sid: source to isocenter distance
// sdd: source to detector distance
__global__ void ForwardProjectionBilinear_device(float* img, float* sgm, const float* u, const float *v, const float* offcenter_array, const float* beta, const float* swing_angle_array, int M, int S,\
	int N, int N_z, int V, float dx, float dz, const float* sid_array, const float* sdd_array, bool conebeam, \
	int z_element_begin_idx, int z_element_end_idx)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;


	if (col < N && row < V && z_element_end_idx <= N_z)
	{
		// half of image side length
		float D = M * dx / 2.0f;
		// half of image thickness
		float D_z = 0.0f;
		if (conebeam)
		{
			D_z = float(S) * dz / 2.0f;
		}
		else
		{
			dz = 0;
		}
		//get the sid and sdd for a given view
		float sid = sid_array[row];
		float sdd = sdd_array[row];

		// current source position
		float xs = sid * cosf(beta[row]);
		float ys = sid * sinf(beta[row]);
		float zs = 0;

		// calculate offcenter bias
		float offcenter_bias = offcenter_array[row] - offcenter_array[0];

		// current detector element position
		float xd = -(sdd - sid) * cosf(beta[row]) + (u[col]+ offcenter_bias) * cosf(beta[row] - PI/2.0f + swing_angle_array[row] /180.0f*PI);
		float yd = -(sdd - sid) * sinf(beta[row]) + (u[col]+ offcenter_bias) * sinf(beta[row] - PI/2.0f + swing_angle_array[row] / 180.0f*PI);
		float zd = 0;

		// step point region
		float L_min = sid - sqrt(2 * D * D + D_z * D_z);
		float L_max = sid + sqrt(2 * D * D + D_z * D_z);

		// source to detector element distance
		float sed = sqrtf((xs - xd)*(xs - xd) + (ys - yd)*(ys - yd));// for fan beam case

		// the point position
		float x, y, z;
		// the point index
		int kx, ky, kz;
		// weighting factor for linear interpolation
		float wx, wy, wz;

		// the most upper left image pixel position
		float x0 = -D + dx / 2.0f;
		float y0 = D - dx / 2.0f;
		float z0 = 0;
		if (conebeam)
		{
			z0 = -D_z + dz / 2.0f;// first slice is at the bottom
		}

		// repeat for each slice
		for (int slice = z_element_begin_idx; slice < z_element_end_idx; slice++)
		{
			// initialization
			//sgm[row*N + col + N * V * slice] = 0;
			sgm[row*N + col] = 0;
			if (conebeam)
			{
				
				zd = v[slice];
				
				sed = sqrtf((xs - xd)*(xs - xd) + (ys - yd)*(ys - yd) + (zs - zd)*(zs - zd));
			}

			// calculate line integration
			for (float L = L_min; L <= L_max; L+= STEPSIZE*sqrt(dx*dx+dz*dz/2.0f))
			{
				// get the current point position 
				x = xs + (xd - xs) * L / sed;
				y = ys + (yd - ys) * L / sed;
				if (conebeam)
				{
					z = zs + (zd - zs) * L / sed;
				}

				// get the current point index
				kx = floorf((x - x0) / dx);
				ky = floorf((y0 - y) / dx);

				if (conebeam)
					kz = floorf((z - z0) / dz);

				// get the image pixel value at the current point
				if(kx>=0 && kx+1<M && ky>=0 && ky+1<M)
				{
					// get the weighting factor
					wx = (x - kx * dx - x0) / dx;
					wy = (y0 - y - ky * dx) / dx;

					// perform bilinear interpolation
					if (conebeam == false)
					{
						sgm[row*N + col] += (1 - wx)*(1 - wy)*img[ky*M + kx + M * M*slice] // upper left
							+ wx * (1 - wy) * img[ky*M + kx + 1 + M * M*slice] // upper right
							+ (1 - wx) * wy * img[(ky + 1)*M + kx + M * M*slice] // bottom left
							+ wx * wy * img[(ky + 1)*M + kx + 1 + M * M*slice];	// bottom right
					}
					else if (conebeam == true && kz >= 0 && kz + 1 < S)
					{
						wz = (z - kz * dz - z0) / dz;
						float sgm_val_lowerslice = (1 - wx)*(1 - wy)*img[ky*M + kx + M * M*kz] // upper left
							+ wx * (1 - wy) * img[ky*M + kx + 1 + M * M*kz] // upper right
							+ (1 - wx) * wy * img[(ky + 1)*M + kx + M * M*kz] // bottom left
							+ wx * wy * img[(ky + 1)*M + kx + 1 + M * M*kz];	// bottom right
						float sgm_val_upperslice = (1 - wx)*(1 - wy)*img[ky*M + kx + M * M*(kz+1)] // upper left
							+ wx * (1 - wy) * img[ky*M + kx + 1 + M * M*(kz + 1)] // upper right
							+ (1 - wx) * wy * img[(ky + 1)*M + kx + M * M*(kz + 1)] // bottom left
							+ wx * wy * img[(ky + 1)*M + kx + 1 + M * M*(kz + 1)];	// bottom right

						sgm[row*N + col] += (1 - wz)*sgm_val_lowerslice + wz * sgm_val_upperslice;
					}
					
				}
			}

			sgm[row*N + col] *= STEPSIZE * sqrt(dx*dx + dz * dz);

		}
	}
}

// sgm_large: sinogram data before binning
// sgm: sinogram data after binning
// N: number of detector elements (after binning)
// V: number of views
// S: number of slices
// binSize: bin size
__global__ void BinSinogram(float* sgm_large, float* sgm, int N, int V, int S, int binSize)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		// repeat for each slice
		for (int slice = 0; slice < S; slice++)
		{
			// initialization
			sgm[row * N + col + N * V * slice] = 0;

			// sum over each bin
			for (int i = 0; i < binSize; i++)
			{
				sgm[row * N + col + N * V * slice] += sgm_large[row * N * binSize + col*binSize + i + slice * N * binSize * V];
			}
			// take average
			sgm[row * N + col + N * V * slice] /= binSize;
		}
	}
}

void InitializeDistance_Agent(float* &distance_array, const float distance, const int V)
{
	if (distance_array != nullptr)
		hipFree(distance_array);

	hipMalloc((void**)&distance_array, V * sizeof(float));
	InitDistance << <(V + 511) / 512, 512 >> > (distance_array, distance, V);
}

void InitializeNonuniformSDD_Agent(float* &distance_array, const int V, const std::string& distanceFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (distance_array != nullptr)
		hipFree(distance_array);

	hipMalloc((void**)&distance_array, V * sizeof(float));

	float* distance_array_cpu = new float[V];
	std::ifstream ifs(distanceFile);
	if (!ifs)
	{
		printf("Cannot find SDD information file '%s'!\n", distanceFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value distance_jsonc_value;
	if (doc.HasMember("SourceDetectorDistance"))
	{

		distance_jsonc_value = doc["SourceDetectorDistance"];

		if (distance_jsonc_value.Size() != V)
		{
			printf("Number of sdd values is %d while the number of Views is %d!\n", distance_jsonc_value.Size(), V);
			exit(-2);
		}

		for (unsigned i = 0; i < distance_jsonc_value.Size(); i++)
		{
			distance_array_cpu[i] = distance_jsonc_value[i].GetFloat();
		}

	}
	else
	{
		printf("Did not find SourceDetectorDistance member in jsonc file!\n");
		exit(-2);
	}

	hipMemcpy(distance_array, distance_array_cpu, sizeof(float)*V, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
}

void InitializeNonuniformSID_Agent(float* &distance_array, const int V, const std::string& distanceFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (distance_array != nullptr)
		hipFree(distance_array);

	hipMallocManaged((void**)&distance_array, V * sizeof(float));
	std::ifstream ifs(distanceFile);
	if (!ifs)
	{
		printf("Cannot find SID information file '%s'!\n", distanceFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value distance_jsonc_value;
	if (doc.HasMember("SourceIsocenterDistance"))
	{

		distance_jsonc_value = doc["SourceIsocenterDistance"];

		if (distance_jsonc_value.Size() != V)
		{
			printf("Number of sid values is %d while the number of Views is %d!\n", distance_jsonc_value.Size(), V);
			exit(-2);
		}

		for (unsigned i = 0; i < distance_jsonc_value.Size(); i++)
		{
			distance_array[i] = distance_jsonc_value[i].GetFloat();
		}

	}
	else
	{
		printf("Did not find SourceIsocenterDistance member in jsonc file!\n");
		exit(-2);
	}

	hipDeviceSynchronize();
}

void InitializeNonuniformOffCenter_Agent(float* &offcenter_array, const int V, const std::string& offCenterFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (offcenter_array != nullptr)
		hipFree(offcenter_array);

	hipMallocManaged((void**)&offcenter_array, V * sizeof(float));
	std::ifstream ifs(offCenterFile);
	if (!ifs)
	{
		printf("Cannot find Offcenter or Swing Angle information file '%s'!\n", offCenterFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value distance_jsonc_value;
	if (doc.HasMember("OffcenterArray"))
	{

		distance_jsonc_value = doc["OffcenterArray"];

		if (distance_jsonc_value.Size() != V)
		{
			printf("Number of offcenter values is %d while the number of Views is %d!\n", distance_jsonc_value.Size(), V);
			exit(-2);
		}

		for (unsigned i = 0; i < distance_jsonc_value.Size(); i++)
		{
			offcenter_array[i] = distance_jsonc_value[i].GetFloat();
		}

	}
	else
	{
		printf("Did not find OffcenterArray member in jsonc file!\n");
		exit(-2);
	}

	hipDeviceSynchronize();
}

//new function with Value member to suit all non uniform parameters
void InitializeNonuniformPara_Agent(float* &para_array, const int V, const std::string& paraFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (para_array != nullptr)
		hipFree(para_array);

	hipMalloc((void**)&para_array, V * sizeof(float));
	float* para_array_cpu = new float[V];

	std::ifstream ifs(paraFile);
	if (!ifs)
	{
		printf("Cannot find file '%s'!\n", paraFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value array_jsonc_value;
	if (doc.HasMember("Value"))
	{
		array_jsonc_value = doc["Value"];
	}
	else
	{
		printf("Did not find Value member in jsonc file!\n");
		exit(-2);
	}


	if (array_jsonc_value.Size() != V)
	{
		printf("Number of elements in the array is %d while the number of Views is %d!\n", array_jsonc_value.Size(), V);
		exit(-2);
	}

	for (unsigned i = 0; i < array_jsonc_value.Size(); i++)
	{
		para_array_cpu[i] = array_jsonc_value[i].GetFloat(); //printf("%d: %f\n", i, para_array_cpu[i]);
	}
	hipMemcpy(para_array, para_array_cpu, sizeof(float)*V, hipMemcpyHostToDevice);
	//printf("copy finished!\n");
	hipDeviceSynchronize();
}

void InitializeU_Agent(float* &u, const int N, const float du, const float offcenter)
{
	if (u != nullptr)
		hipFree(u);

	hipMalloc((void**)&u, N * sizeof(float));
	InitU <<<(N + 511) / 512, 512 >>> (u, N, du, offcenter);
}

void InitializeBeta_Agent(float *& beta, const int V, const float startAngle, const float totalScanAngle)
{
	if (beta != nullptr)
		hipFree(beta);

	hipMalloc((void**)&beta, V * sizeof(float));
	InitBeta <<< (V + 511) / 512, 512 >>> (beta, V, startAngle, totalScanAngle);
}

void InitializeNonuniformBeta_Agent(float* &beta, const int V, const float rotation, const std::string& scanAngleFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (beta != nullptr)
		hipFree(beta);

	hipMalloc((void**)&beta, V * sizeof(float));
	float * beta_cpu = new float[V];
	std::ifstream ifs(scanAngleFile);
	if (!ifs)
	{
		printf("Cannot find angle information file '%s'!\n", scanAngleFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value scan_angle_jsonc_value;
	if (doc.HasMember("ScanAngle"))
	{
		scan_angle_jsonc_value = doc["ScanAngle"];
	}
	else if (doc.HasMember("Value"))
	{
		scan_angle_jsonc_value = doc["Value"];
	}
	else
	{
		printf("Did not find ScanAngle or Value member in jsonc file!\n");
		exit(-2);
	}

	if (scan_angle_jsonc_value.Size() != V)
	{
		printf("Number of scan angles is %d while the number of Views is %d!\n", scan_angle_jsonc_value.Size(), V);
		exit(-2);
	}

	for (unsigned i = 0; i < scan_angle_jsonc_value.Size(); i++)
	{
		beta_cpu[i] = rotation / 180.0f*PI + scan_angle_jsonc_value[i].GetFloat() / 180.0*PI;
	}
	hipMemcpy(beta, beta_cpu, sizeof(float)*V, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void ForwardProjectionBilinear_Agent(float *& image, float * &sinogram, const float* sid_array, const float* sdd_array, const float* offcenter_array,\
	const float* u, const float *v, const float* beta, const float* swing_angle_array, const mango::Config & config, int z_element_idx)
{
	dim3 grid((config.detEltCount*config.oversampleSize + 7) / 8, (config.views + 7) / 8);
	dim3 block(8, 8);

	ForwardProjectionBilinear_device<<<grid, block>>>(image, sinogram, u, v, offcenter_array, beta, swing_angle_array, config.imgDim, config.sliceCount,\
		config.detEltCount*config.oversampleSize, config.detZEltCount, config.views, config.pixelSize, config.sliceThickness, sid_array, sdd_array, config.coneBeam, z_element_idx, z_element_idx+1);

	hipDeviceSynchronize();
}

void BinSinogram(float* &sinogram_large, float* &sinogram, const mango::Config& config)
{
	dim3 grid((config.detEltCount + 7) / 8, (config.views + 7) / 8);
	dim3 block(8, 8);
	
	BinSinogram <<<grid, block >>> (sinogram_large, sinogram, config.detEltCount, config.views, 1, config.oversampleSize);
	// since the sinogram has only one slice, the z_element count is 1

	hipDeviceSynchronize();
}

void SaveSinogramSlice(const char * filename, float*&sinogram_slice, int z_element_idx, const mango::Config& config)
{
	FILE* fp = NULL;
	if (z_element_idx == 0)
		fp = fopen(filename, "wb");
	else
		fp = fopen(filename, "ab");

	if (fp == NULL)
	{
		fprintf(stderr, "Cannot save to file %s!\n", filename);
		exit(4);
	}
	fwrite(sinogram_slice, sizeof(float), config.detEltCount * config.views, fp);
	fclose(fp);
}

void MallocManaged_Agent(float * &p, const int size)
{
	hipMallocManaged((void**)&p, size);
}

void FreeMemory_Agent(float* &p)
{
	hipFree(p);
	p = nullptr;
}

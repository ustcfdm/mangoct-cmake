#include "hip/hip_runtime.h"
#include "FbpClass_Agent.cuh"
#include <stdio.h>
#include "stdafx.h"
#define PI 3.1415926536f


__global__ void InitDistance(float *distance_array, const float distance, const int V)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < V)
	{
		distance_array[tid] = distance;
	}
}

__global__ void InitU(float* u, const int N, const float du, const float offcenter)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < N)
	{
		u[tid] = (tid - (N - 1) / 2.0f) * du + offcenter;
	}
}

__global__ void InitBeta(float* beta, const int V, const float rotation, const float totalScanAngle)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < V)
	{
		beta[tid] = (totalScanAngle / V * tid + rotation) * PI / 180;
	}
}

__global__ void InitReconKernel_Hamming(float* reconKernel, const int N, const float du, const float t)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < 2 * N - 1)
	{
		// the center element index is N-1
		int n = tid - (N - 1);

		// ramp part
		if (n == 0)
			reconKernel[tid] = t / (4 * du*du);
		else if (n % 2 == 0)
			reconKernel[tid] = 0;
		else
			reconKernel[tid] = -t / (n*n * PI*PI * du*du);

		// cosine part
		int sgn = n % 2 == 0 ? 1 : -1;

		reconKernel[tid] += (1 - t)* (sgn / (2 * PI*du*du) * (1.0f / (1 + 2 * n) + 1.0f / (1 - 2 * n))
			- 1 / (PI*PI*du*du) * (1.0f / (1 + 2 * n) / (1 + 2 * n) + 1.0f / (1 - 2 * n) / (1 - 2 * n)));
	}
}

__global__ void InitReconKernel_Delta(float* reconKernel, const int N, const float du, const float t)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < 2 * N - 1)
	{
		// the center element index is N-1
		int n = tid - (N - 1);

		if (n == 0)
			reconKernel[tid] = t;
		else
			reconKernel[tid] = 0;
	}
}

//initialize a Gaussian kernel
//This kernel will be used along with the ramp kernel
//delta is in number of pixels, which is the standard deviation of the gaussian
//This kernel is normalized
__global__ void InitReconKernel_GaussianApodized(float* reconKernel, const int N, const float du, const float delta)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < 1)
	{
		// the center element index is N-1
		float temp_sum = 0;
		for (int i = 0; i < 2 * N - 1; i++)
		{
			int n = i - (N - 1);
			reconKernel[i] = exp( - float(n) * float(n) / 2.0 / delta / delta);
			temp_sum = temp_sum + reconKernel[i];
		}

		for (int i = 0; i < 2 * N - 1; i++)
		{
			reconKernel[i] = reconKernel[i] / temp_sum / du;
		}
	}
}


__global__ void InitReconKernel_Quadratic(float* reconKernel, const int N, const float du, const int paramNum, const float p1, const float p2, const float p3)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < 2 * N - 1)
	{
		float a, b, c;

		float kn = 1 / (2 * du);

		if (paramNum == 2)
		{
			// p1 = t, p2 = h, p3 is ignored
			a = (p2 - 1) / (kn*kn * (1 - 2 * p1));
			b = -2 * a*p1*kn;
			c = 1.0f;
		}
		else
		{
			a = p1;
			b = p2;
			c = p3;
		}

		reconKernel[idx] = 0.0f;

		float du2 = du * du;
		float du3 = du2 * du;
		float du4 = du3 * du;

		int n = idx - (N - 1);
		if (n == 0)
		{
			// H3(x)
			reconKernel[idx] += a / 32 / du4;
			// H2(x)
			reconKernel[idx] += b / 12 / du3;
			// H1(x)
			reconKernel[idx] += c / 4 / du2;
		}
		else if (n % 2 == 0)
		{
			// H3(x)
			reconKernel[idx] += a * 3 / (8 * n*n * PI*PI * du4);
			// H2(x)
			reconKernel[idx] += b / (2 * n*n * PI*PI * du3);
			// H1(x)
			// do nothing, H1(even) is zero
		}
		else
		{
			// H3(x)
			reconKernel[idx] += a * 3 / (8 * n*n * PI*PI * du4) *  (4 / (n*n*PI*PI) - 1);
			// H2(x)
			reconKernel[idx] += -b / (2 * n*n * PI*PI * du3);
			// H1(x)
			reconKernel[idx] += -c / (n*n * PI*PI * du2);
		}
	}
}

__global__ void InitReconKernel_Polynomial(float* reconKernel, const int N, const float du, const float p6, const float p5, const float p4, const float p3, const float p2, const float p1, const float p0)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < 2 * N - 1)
	{
		int n = idx - (N - 1);
		reconKernel[idx] = 0.0f;
		float kn = 1 / (2 * du);

		float du2 = du * du;
		float du3 = du2 * du;
		float du4 = du3 * du;

		if (n == 0)
		{
			// H7(x)
			reconKernel[idx] += p6 * powf(kn, 8) / 4;
			// H6(x)
			reconKernel[idx] += p5 * powf(kn, 7) * 2 / 7;
			// H5(x)
			reconKernel[idx] += p4 * powf(kn, 6) / 3;
			// H4(x)
			reconKernel[idx] += p3 * powf(kn, 5) * 2 / 5;
			// H3(x)
			reconKernel[idx] += p2 * powf(kn, 4) / 2;
			// H2(x)
			reconKernel[idx] += p1 * 2 * kn*kn*kn / 3;
			// H1(x)
			reconKernel[idx] += p0 * kn*kn;
		}
		else if (n % 2 == 0)
		{
			// H7(x)
			reconKernel[idx] += p6 * 7 * (360 - 30 * n*n*PI*PI + powf(n*PI, 4)) / (128 * du2* powf(du*n*PI, 6));
			// H6(x)
			reconKernel[idx] += p5 * 3 * (120 - 20 * n*n*PI*PI + powf(n*PI, 4)) / (32 * du*powf(du*n*PI, 6));
			// H5(x)
			reconKernel[idx] += p4 * 5 * (n*n*PI*PI - 12) / (32 * du2 *powf(du*n*PI, 4));
			// H4(x)
			reconKernel[idx] += p3 * (n*n*PI*PI - 6) / (4 * du * powf(du*n*PI, 4));
			// H3(x)
			reconKernel[idx] += p2 * 3 / (8 * du4 * n*n * PI*PI);
			// H2(x)
			reconKernel[idx] += p1 / (2 * n*n *PI*PI * du3);
			// H1(x)
			// do nothing, H1(even) is zero
		}
		else
		{
			// H7(x)
			reconKernel[idx] += p6 * 7 * (1440 - 360 * n*n*PI*PI + 30 * powf(n*PI, 4) - powf(n*PI, 6)) / (128 * powf(du*n*PI, 8));
			// H6(x)
			reconKernel[idx] += -p5 * 3 * (120 - 20 * n*n*PI*PI + powf(n*PI, 4)) / (32 * du*powf(du*n*PI, 6));
			// H5(x)
			reconKernel[idx] += -p4 * 5 * (48 - 12 * n*n*PI*PI + powf(n*PI, 4)) / (32 * powf(du*n*PI, 6));
			// H4(x)
			reconKernel[idx] += p3 * (6 - n * n*PI*PI) / (4 * du * powf(du*n*PI, 4));
			// H3(x)
			reconKernel[idx] += p2 * (4 - n * n*PI*PI) * 3 / (8 * powf(du*n*PI, 4));
			// H2(x)
			reconKernel[idx] += -p1 / (2 * n*n *PI*PI * du3);
			// H1(x)
			reconKernel[idx] += -p0 / (n*n *PI*PI * du2);
		}
	}
}

__global__ void InitReconKernel_Hilbert(float* reconKernel, const int N, const float du, const float t)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < 2 * N - 1)
	{
		int n = tid - (N - 1);

		if (n % 2 == 0)
			reconKernel[tid] = 0;
		else
		{
			reconKernel[tid] = 1 / (PI * PI * n * du);
			if (t < 0)
				reconKernel[tid] = -reconKernel[tid];
		}
	}
}

// weight the sinogram data
// sgm: sinogram (width x height x slice)
// N: width
// H: height
// V: views
// S: slice
// sliceThickness: mm
// sliceOffcenter: mm
// sdd: source to detector distance
// totalScanAngle
__global__ void WeightSinogram_device(float* sgm, const float* u, const int N, const int H, const int V, \
	const int S, const float sliceThickness, const float sliceOffcenter, float* sdd_array, float totalScanAngle, bool shortScan, float *beta_array, float* offcenter_array)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		float offcenter_bias = offcenter_array[row] - offcenter_array[0];
		float u_actual = u[col] + offcenter_bias;//actual u value due to non uniform offcenter

		float sdd = sdd_array[row];
		for (int i = 0; i < S; i++)
		{
			float v = sliceThickness * (i - (float(S) / 2.0f + 0.5)) + sliceOffcenter;
			sgm[row*N + col + i * N*H] *= sdd * sdd / sqrtf((u_actual)*(u_actual)+sdd * sdd + v * v);
			//the loop is to include all the slices since there may be more than one slice
		}

		if (shortScan)
		{
			//this beta is different from the beta_array
			//To calculate the parker weighting, beta should begin with zero degree
			//while the betaArray includes the start rotation angle

			//adding abs function to deal with the case when totalScanAngle is negative
			float beta = abs(beta_array[row] - beta_array[0]);
			float rotation_direction = abs(totalScanAngle) / (totalScanAngle);
			float gamma = atan(u_actual / sdd) * rotation_direction;

			//float beta = abs(totalScanAngle/ 180.0f * PI ) / float(V) * float(row) ;
			//float gamma = abs(totalScanAngle) / totalScanAngle * atan(u[col] / sdd);
			float gamma_max = abs(totalScanAngle) * PI / 180.0f - PI;

			//calculation of the parker weighting
			float weighting = 0;
			if (beta >= 0 && beta < gamma_max - 2 * gamma)
			{
				weighting = sin(PI / 2 * beta / (gamma_max - 2 * gamma));
				weighting = weighting * weighting;
			}
			else if (beta >= gamma_max - 2 * gamma && beta < PI - 2 * gamma)
			{
				weighting = 1;
			}
			else if (beta >= PI - 2 * gamma && beta <= PI + gamma_max)
			{
				weighting = sin(PI / 2 * (PI + gamma_max - beta) / (gamma_max + 2 * gamma));
				weighting = weighting * weighting;
			}
			else
			{
				//printf("ERROR!");
			}
			for (int i = 0; i < S; i++)
			{
				sgm[row*N + col + i * N*H] *= weighting;
			}
		}
		else
		{
			;
		}
	}

}


// weight the sinogram data of Hilbert kernel (for phase contrast imaging)
// sgm: sinogram (width x height x slice)
// N: width
// V: height (views)
// S: slice
// sdd: source to detector distance
__global__ void WeightSinogramHilbert_device(float* sgm, const float* u, const int N, const int V, const int S, float sdd)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		for (int i = 0; i < S; i++)
		{
			sgm[row*N + col + i * N*V] *= sqrtf(u[col] * u[col] + sdd * sdd);
		}
	}
}


// weight the sinogram data of Hilbert kernel (for phase contrast imaging) along angle direction (temporary test)
// sgm: sinogram (width x height x slice)
// N: width
// V: height (views)
// S: slice
// sdd: source to detector distance
__global__ void WeightSinogramHilbert_angle_device(float* sgm, const float* u, const int N, const int V, const int S, float sdd)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		for (int i = 0; i < S; i++)
		{
			sgm[row*N + col + i * N*V] *= sdd / sqrtf(u[col] * u[col] + sdd * sdd);
		}
	}
}

// perform beam hardening correction of sinogram
// sgm: sinogram (width x height x slice)
// N: width
// V: height (views)
// S: slice
// p0-p9: correction parameters
__global__ void CorrectBeamHardening_device(float* sgm, const int N, const int V, const int S, float p0, float p1, float p2, float p3, float p4, float p5, float p6, float p7, float p8, float p9)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		for (int i = 0; i < S; i++)
		{
			float oldSgm = sgm[row*N + col + i * N*V];
			sgm[row*N + col + i * N*V] = p0 + p1 * powf(oldSgm, 1) + p2 * powf(oldSgm, 2) + p3 * powf(oldSgm, 3) + p4 * powf(oldSgm, 4) + p5 * powf(oldSgm, 5) + p6 * powf(oldSgm, 6) + p7 * powf(oldSgm, 7) + p8 * powf(oldSgm, 8) + p9 * powf(oldSgm, 9);
		}
	}
}

// convolve the sinogram data
// sgm_flt: sinogram data after convolving
// sgm: initial sinogram data
// reconKernel: reconstruction kernel
// N: sinogram width
// H: sinogram height
// V: number of views
// S: number of slices
// u: the position (coordinate) of each detector element
// du: detector element size [mm]
__global__ void ConvolveSinogram_device(float* sgm_flt, const float* sgm, float* reconKernel, const int N, const int H, const int V, const int S, const float* u, const float du)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		for (int slice = 0; slice < S; slice++)
		{
			// temporary variable to speed up
			float sgm_flt_local = 0;

			for (int i = 0; i < N; i++)
			{
				sgm_flt_local += sgm[row*N + i + slice * N*H] * reconKernel[N - 1 - col + i];
			}
			sgm_flt[row*N + col + slice * N*V] = sgm_flt_local * du;
		}

	}
}

// Copy the sinogram data from one array(pointer) to another array(pointer). This is for "None" kernel reconstruction.
// sgm_flt: sinogram data after copy
// sgm: initial sinogram data
// N: sinogram width
// H: sinogram height
// V: number of views
// S: number of slices
__global__ void CopySinogram_device(float* sgm_flt, const float* sgm, const int N, const int H, const int V, const int S)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		for (int slice = 0; slice < S; slice++)
		{
			sgm_flt[row * N + col + slice * N * V] = sgm[row * N + col + slice * N * V];
		}
	}
}

// backproject the image using pixel-driven method
// sgm: sinogram data
// img: image data
// U: each detector element position [mm]
// u: detector pixel array
// v: detector pixel array in z axis
// beta: view angle [radius]
// N: number of detector elements
// V: number of views
// totalScanAngle [rads]
// S: number of slices of the sinogram
// coneBeam: whether the recon is conbeam recon or not
// M: image dimension
// imgS: image slice count
// sdd: source to detector distance [mm]
// sid: source to isocenter distance [mm]
// du: detector element size [mm]
// dv: detector element height [mm]
// dx: image pixel size [mm]
// dz: image slice thickness [mm]
// (xc, yc, zc): image center position [mm, mm, mm]
__global__ void BackprojectPixelDriven_device(float* sgm, float* img, float* u, float* v, float* beta, bool shortScan, const int N, const int V, \
	const int S, bool coneBeam, const int M, const int imgS, float* sdd_array, float* sid_array, float* offcenter_array, const float dx, const float dz, \
	const float xc, const float yc, const float zc, int imgS_idx)
{

	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	float du = u[1] - u[0];
	float dv = v[1] - v[0];


	if (col < M && row < M && imgS_idx <= imgS)
	{

		float x = (col - (M - 1) / 2.0f)*dx + xc;
		float y = ((M - 1) / 2.0f - row)*dx + yc;

		float z;

		float U, u0, v0;
		float mag_factor;
		float w;
		int k;

		float w_z;//weight for cbct
		int k_z;//index for cbct
		float delta_beta;// delta_beta for the integral calculation (nonuniform scan angle)

		float lower_row_val, upper_row_val;

		for (int slice = imgS_idx; slice < imgS_idx + 1; slice++)
		{

			z = (slice - (float(imgS) - 1.0f) / 2.0f) * dz + zc;

			// temporary local variable to speed up
			float img_local = 0;

			for (int view = 0; view < V; view++)
			{
				float offcenter_bias = offcenter_array[view] - offcenter_array[0];
				float sid = sid_array[view];
				float sdd = sdd_array[view];
				//calculation of delta_beta for the integral calculation
				if (view == 0)
					delta_beta = abs(beta[1] - beta[0]);
				else if (view == V - 1)
					delta_beta = abs(beta[view] - beta[view - 1]);
				else
					delta_beta = abs(beta[view + 1] - beta[view - 1]) / 2.0f;

				U = sid - x * cosf(beta[view]) - y * sinf(beta[view]);

				//calculate the magnification
				mag_factor = sdd / U;

				// find u0 
				u0 = mag_factor * (x*sinf(beta[view]) - y * cosf(beta[view]));


				k = floorf((u0 - (u[0] + offcenter_bias)) / du);
				if (k<0 || k + 1>N - 1)
				{
					img_local = 0;
					break;
				}

				w = (u0 - (u[k] + offcenter_bias)) / du;

				// for cone beam ct, we also need to find v0
				if (coneBeam && abs(dv) > 0.00001f)
				{
					v0 = mag_factor * z;
					// weight for cbct recon
					k_z = floorf((v0 - v[0]) / dv);
					if (k_z<0 || k_z + 1>S - 1)
					{
						img_local = 0;
						break;
					}

					w_z = (v0 - v[k_z]) / dv;

					lower_row_val = (w*sgm[view*N + k + 1 + k_z * N*V] + (1 - w)*sgm[view*N + k + k_z * N*V]);
					upper_row_val = (w*sgm[view*N + k + 1 + (k_z + 1) * N*V] + (1 - w)*sgm[view*N + k + (k_z + 1) * N*V]);

					img_local += sid / U / U * (w_z*upper_row_val + (1 - w_z)*lower_row_val) * delta_beta;
				}

				else
				{
					img_local += sid / U / U * (w*sgm[view*N + k + 1 + slice * N*V] + (1 - w)*sgm[view*N + k + slice * N*V]) * delta_beta;
				}
			}

			//judge whether the scan is a full scan or a short scan
			if (shortScan)
			{
				//printf("this is a full scan");
				img[row*M + col] = img_local;

			}
			else
				img[row*M + col] = img_local / 2.0f;


		}
	}
}

__global__ void BackprojectPixelDriven_pmatrix_device(float* sgm, float* img, float* u, float* v, float* beta, float* pmatrix, \
	bool shortScan, const int N, const int V, const int S, bool coneBeam, const int M, const int imgS, float* sdd_array, float* sid_array, \
	const float dx, const float dz, const float xc, const float yc, const float zc, int imgS_idx, float imgRot)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	float du = u[1] - u[0];
	float imgRot_in_rad = imgRot * PI / 180.0f;
	if (col < M && row < M && imgS_idx < imgS)
	{

		float x_after_rotation = (col - (M - 1) / 2.0f)*dx + xc;
		float y_after_rotation = ((M - 1) / 2.0f - row)*dx + yc;

		

		float x = x_after_rotation * cos(imgRot_in_rad) + y_after_rotation * sin(imgRot_in_rad);//(col - (M - 1) / 2.0f)*dx + xc;
		float y = y_after_rotation * cos(imgRot_in_rad) - x_after_rotation * sin(imgRot_in_rad);//((M - 1) / 2.0f - row)*dx + yc;
		float z;
		float U;
		float w;
		int k;
		float w_z;//weight for cbct
		int k_z;//index for cbct
		float delta_beta;// delta_beta for the integral calculation (nonuniform scan angle)

		float lower_row_val, upper_row_val;

		for (int slice = imgS_idx; slice < imgS_idx + 1; slice++)
		{

			z = (slice - (float(imgS) - 1.0f) / 2.0f) * dz + zc;

			// temporary local variable to speed up
			float img_local = 0;

			for (int view = 0; view < V; view++)
			{
				float sid = sid_array[view];
				float sdd = sdd_array[view];
				//calculation of delta_beta for the integral calculation
				if (view == 0)
					delta_beta = abs(beta[1] - beta[0]);
				else if (view == V - 1)
					delta_beta = abs(beta[view] - beta[view - 1]);
				else
					delta_beta = abs(beta[view + 1] - beta[view - 1]) / 2.0f;



				//use pmatrix to calculate the corresponding index on the detector
				int pos_in_matrix = 12 * view;
				float k_u_divide_mag = pmatrix[pos_in_matrix] * x + pmatrix[pos_in_matrix + 1] * y + pmatrix[pos_in_matrix + 2] * z + pmatrix[pos_in_matrix + 3] * 1;
				float one_divide_mag = pmatrix[pos_in_matrix + 8] * x + pmatrix[pos_in_matrix + 9] * y + pmatrix[pos_in_matrix + 10] * z + pmatrix[pos_in_matrix + 11] * 1;
				
				//the pmatrix is calculated when the detector is binned with 4 pixels
				// each after binning is 0.4 mm
				float k_f_bin_4 = k_u_divide_mag / one_divide_mag;//float number of k_f_bin_4
				float u_position_true = (k_f_bin_4 + 0.5f)*0.4;
				float k_f = u_position_true / du - 0.5f;

				//float k_f = k_u_divide_mag / one_divide_mag;//float number of k
				k = floorf(k_f);

				//the pmatrix is acquired assuming beta[0]=0
				//however, in a real recon, the image need to be rotated
				//we need to retrieve the beta value for the pmatrix recon
				//for calculation of U
				float beta_pmatrix = beta[view] - beta[0];
				U = sid - x * cosf(beta_pmatrix) - y * sinf(beta_pmatrix);

				if (k<0 || k + 1>N - 1)
				{
					img_local = 0;
					break;
				}

				w = k_f - k;

				// for cone beam ct, we also need to find v0
				if (coneBeam)
				{
					float k_z_divide_mag = pmatrix[pos_in_matrix + 4] * x + pmatrix[pos_in_matrix + 5] * y + pmatrix[pos_in_matrix + 6] * z + pmatrix[pos_in_matrix + 7] * 1;
					float k_z_f = k_z_divide_mag / one_divide_mag;//float number of k_z
					k_z = floorf(k_z_f);
					if (k_z<0 || k_z + 1>S - 1)
					{
						img_local = 0;
						break;
					}

					w_z = k_z_f - k_z;

					lower_row_val = (w*sgm[view*N + k + 1 + k_z * N*V] + (1 - w)*sgm[view*N + k + k_z * N*V]);
					upper_row_val = (w*sgm[view*N + k + 1 + (k_z + 1) * N*V] + (1 - w)*sgm[view*N + k + (k_z + 1) * N*V]);

					img_local += sid / U / U * (w_z*upper_row_val + (1 - w_z)*lower_row_val) * delta_beta;
				}

				else
				{
					img_local += sid / U / U * (w*sgm[view*N + k + 1 + slice * N*V] + (1 - w)*sgm[view*N + k + slice * N*V]) * delta_beta;
				}
			}

			//judge whether the scan is a full scan or a short scan
			if (shortScan)
			{
				//printf("this is a full scan");
				img[row*M + col] = img_local;

			}
			else
			{
				img[row*M + col] = img_local / 2.0f;
			}


		}
	}
}


// backproject the image using pixel-driven method for Hilbert kernel (for phase contrast imaging)
// sgm: sinogram data
// img: image data
// U: each detector element position [mm]
// beta: view angle [radius]
// N: number of detector elements
// V: number of views
// S: number of slices
// M: image dimension
// sdd: source to detector distance [mm]
// sid: source to isocenter distance [mm]
// du: detector element size [mm]
// dx: image pixel size [mm]
// (xc, yc): image center position [mm, mm]
__global__ void BackprojectPixelDrivenHilbert_device(float* sgm, float* img, float* u, float* beta, const int N, const int V, \
	const int S, const int M, const float sdd, const float sid, const float du, const float dx, const float xc, const float yc, int imgS_idx)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < M && row < M && imgS_idx < S)
	{
		float x = (col - (M - 1) / 2.0f)*dx + xc;
		float y = ((M - 1) / 2.0f - row)*dx + yc;

		float U, u0;
		float w;
		int k;

		for (int slice = imgS_idx; slice < imgS_idx + 1; slice++)
		{
			img[row*M + col] = 0;

			for (int view = 0; view < V; view++)
			{
				U = sid - x * cosf(beta[view]) - y * sinf(beta[view]);
				u0 = sdd * (x*sinf(beta[view]) - y * cosf(beta[view])) / U;

				k = floorf((u0 - u[0]) / du);
				if (k<0 || k + 1>N - 1)
				{
					img[row*M + col] = 0;
					break;
				}

				w = (u0 - u[k]) / du;

				img[row*M + col] += 1 / U * (w*sgm[view*N + k + 1 + slice * N*V] + (1 - w)*sgm[view*N + k + slice * N*V]);

			}
			img[row*M + col] *= PI / V;
		}
	}
}


void InitializeDistance_Agent(float* &distance_array, const float distance, const int V)
{
	if (distance_array != nullptr)
		hipFree(distance_array);

	hipMalloc((void**)&distance_array, V * sizeof(float));
	InitDistance << <(V + 511) / 512, 512 >> > (distance_array, distance, V);
}

void InitializeNonuniformSDD_Agent(float* &distance_array, const int V, const std::string& distanceFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (distance_array != nullptr)
		hipFree(distance_array);

	hipMalloc((void**)&distance_array, V * sizeof(float));
	float* distance_array_cpu = new float[V];
	std::ifstream ifs(distanceFile);
	if (!ifs)
	{
		printf("\nCannot find SDD information file '%s'!\n", distanceFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value distance_jsonc_value;
	if (doc.HasMember("SourceDetectorDistance"))
	{

		distance_jsonc_value = doc["SourceDetectorDistance"];

	} 
	else if(doc.HasMember("Value"))//a new version of the program uses value as member to avoid complex member names
	{
		distance_jsonc_value = doc["Value"];
	}
	else
	{
		printf("\nDid not find Value member in jsonc file!\n");
		exit(-2);
	}

	if (distance_jsonc_value.Size() != V)
	{
		printf("\nNumber of sdd values is %d while the number of Views is %d!\n", distance_jsonc_value.Size(), V);
		exit(-2);
	}

	for (unsigned i = 0; i < distance_jsonc_value.Size(); i++)
	{
		distance_array_cpu[i] = distance_jsonc_value[i].GetFloat();
	}

	hipMemcpy(distance_array, distance_array_cpu, V * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void InitializeNonuniformSID_Agent(float* &distance_array, const int V, const std::string& distanceFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (distance_array != nullptr)
		hipFree(distance_array);

	hipMalloc((void**)&distance_array, V * sizeof(float));
	float* distance_array_cpu = new float[V];
	std::ifstream ifs(distanceFile);
	if (!ifs)
	{
		printf("\nCannot find SID information file '%s'!\n", distanceFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value distance_jsonc_value;
	if (doc.HasMember("SourceIsocenterDistance"))
	{
		distance_jsonc_value = doc["SourceIsocenterDistance"];
	}
	else if (doc.HasMember("Value"))//a new version of the program uses value as member to avoid complex member names
	{
		distance_jsonc_value = doc["Value"];
	}
	else
	{
		printf("\nDid not find SourceIsocenterDistance or Value member in jsonc file!\n");
		exit(-2);
	}

	if (distance_jsonc_value.Size() != V)
	{
		printf("\nNumber of sid values is %d while the number of Views is %d!\n", distance_jsonc_value.Size(), V);
		exit(-2);
	}

	for (unsigned i = 0; i < distance_jsonc_value.Size(); i++)
	{
		distance_array_cpu[i] = distance_jsonc_value[i].GetFloat();
	}

	hipMemcpy(distance_array, distance_array_cpu, V * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void InitializeNonuniformOffCenter_Agent(float* &distance_array, const int V, const std::string& distanceFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (distance_array != nullptr)
		hipFree(distance_array);

	hipMalloc((void**)&distance_array, V * sizeof(float));
	float* distance_array_cpu = new float[V];
	std::ifstream ifs(distanceFile);
	if (!ifs)
	{
		printf("\nCannot find Offcenter information file '%s'!\n", distanceFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value distance_jsonc_value;
	if (doc.HasMember("OffcenterArray"))
	{
		distance_jsonc_value = doc["OffcenterArray"];
	}
	else if (doc.HasMember("Value"))//a new version of the program uses value as member to avoid complex member names
	{
		distance_jsonc_value = doc["Value"];
	}
	else
	{
		printf("\nDid not find OffcenterArray or Value member in jsonc file!\n");
		exit(-2);
	}

	if (distance_jsonc_value.Size() != V)
	{
		printf("\nNumber of offcenter values is %d while the number of Views is %d!\n", distance_jsonc_value.Size(), V);
		exit(-2);
	}

	for (unsigned i = 0; i < distance_jsonc_value.Size(); i++)
	{
		distance_array_cpu[i] = distance_jsonc_value[i].GetFloat();
	}

	hipMemcpy(distance_array, distance_array_cpu, V * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void InitializePMatrix_Agent(float* &pmatrix_array, const int V, const std::string& pmatrixFile)
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (pmatrix_array != nullptr)
		hipFree(pmatrix_array);

	//hipMallocManaged((void**)&pmatrix_array, 12 * V * sizeof(float));
	hipMalloc((void**)&pmatrix_array, 12 * V * sizeof(float));
	//hipMallocManaged somehow does not work for this function
	//so hipMalloc and hipMemcpy is used


	float* pmatrix_array_cpu = new float[12 * V];


	std::ifstream ifs(pmatrixFile);
	if (!ifs)
	{
		printf("\nCannot find pmatrix information file '%s'!\n", pmatrixFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value pmatrix_jsonc_value;
	if (doc.HasMember("PMatrix"))
	{
		pmatrix_jsonc_value = doc["PMatrix"];
	}
	else if(doc.HasMember("Value"))
	{
		pmatrix_jsonc_value = doc["Value"];
	}
	else
	{
		printf("\nDid not find PMatrix or Value member in jsonc file!\n");
		exit(-2);
	}
	if (pmatrix_jsonc_value.Size() != 12 * V)
	{
		printf("\nNumber of pmatrix elements is %d while the 12 times number of Views is %d!\n", pmatrix_jsonc_value.Size(), 12 * V);
		exit(-2);
	}

	for (unsigned i = 0; i < 12 * V; i++)
	{
		//printf("\n%d: %f",i, pmatrix_jsonc_value[i].GetFloat());
		pmatrix_array_cpu[i] = pmatrix_jsonc_value[i].GetFloat();
	}
	hipMemcpy(pmatrix_array, pmatrix_array_cpu, 12 * V * sizeof(float), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
}


void InitializeU_Agent(float* &u, const int N, const float du, const float offcenter)
{
	if (u != nullptr)
		hipFree(u);

	hipMalloc((void**)&u, N * sizeof(float));
	InitU << <(N + 511) / 512, 512 >> > (u, N, du, offcenter);
}

void InitializeBeta_Agent(float* &beta, const int V, const float rotation, const float totalScanAngle)
{
	if (beta != nullptr)
		hipFree(beta);

	hipMalloc((void**)&beta, V * sizeof(float));
	InitBeta << < (V + 511) / 512, 512 >> > (beta, V, rotation, totalScanAngle);
}

void InitializeNonuniformBeta_Agent(float* &beta, const int V, const float rotation, const std::string& scanAngleFile)
//unit of beta is RADs
{
	namespace fs = std::filesystem;
	namespace js = rapidjson;

	if (beta != nullptr)
		hipFree(beta);

	hipMalloc((void**)&beta, V * sizeof(float));
	float* beta_cpu = new float[V];
	std::ifstream ifs(scanAngleFile);
	if (!ifs)
	{
		printf("Cannot find angle information file '%s'!\n", scanAngleFile.c_str());
		exit(-2);
	}
	rapidjson::IStreamWrapper isw(ifs);
	rapidjson::Document doc;
	doc.ParseStream<js::kParseCommentsFlag | js::kParseTrailingCommasFlag>(isw);
	js::Value scan_angle_jsonc_value;
	if (doc.HasMember("ScanAngle"))
	{
		scan_angle_jsonc_value = doc["ScanAngle"];
	}
	else if (doc.HasMember("Value"))
	{
		scan_angle_jsonc_value = doc["Value"];
	}
	else
	{
		printf("Did not find ScanAngle or Value member in jsonc file!\n");
		exit(-2);
	}

	if (scan_angle_jsonc_value.Size() != V)
	{
		printf("Number of scan angles is %d while the number of Views is %d!\n", scan_angle_jsonc_value.Size(), V);
		exit(-2);
	}

	for (unsigned i = 0; i < scan_angle_jsonc_value.Size(); i++)
	{
		beta_cpu[i] = rotation / 180.0f*PI + scan_angle_jsonc_value[i].GetFloat() / 180.0*PI;
	}
	hipMemcpy(beta, beta_cpu, sizeof(float)*V, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void InitializeReconKernel_Agent(float* &reconKernel, const int N, const float du, const std::string& kernelName, const std::vector<float>& kernelParam)
{
	if (reconKernel != nullptr)
		hipFree(reconKernel);

	hipMalloc((void**)&reconKernel, (2 * N - 1) * sizeof(float));

	if (kernelName == "HammingFilter")
	{
		InitReconKernel_Hamming << <(2 * N - 1 + 511) / 512, 512 >> > (reconKernel, N, du, kernelParam[0]);
	}
	if (kernelName == "Delta")
	{
		InitReconKernel_Delta << <(2 * N - 1 + 511) / 512, 512 >> > (reconKernel, N, du, kernelParam[0]);
	}
	else if (kernelName == "QuadraticFilter")
	{
		float lastParam = 0.0f;
		if (kernelParam.size() == 3)
			lastParam = kernelParam[2];

		InitReconKernel_Quadratic << <(2 * N - 1 + 511) / 512, 512 >> > (reconKernel, N, du, int(kernelParam.size()), kernelParam[0], kernelParam[1], lastParam);
	}
	else if (kernelName == "Polynomial")
	{
		// TODO: 
		// InitReconKernel_Polynomial <<<...>>> (...);
		float p[7] = { 0 };

		for (size_t i = 0; i < kernelParam.size(); i++)
		{
			p[i] = kernelParam[kernelParam.size() - 1 - i];
		}

		//InitReconKernel_Polynomial <<<(2 * N - 1 + 511) / 512, 512>>> (reconKernel, N, du, p[0], p[1], p[2], p[3], p[4], p[5], p[6]);
		InitReconKernel_Polynomial << <(2 * N - 1 + 511) / 512, 512 >> > (reconKernel, N, du, p[6], p[5], p[4], p[3], p[2], p[1], p[0]);
	}
	else if (kernelName == "Hilbert" || kernelName == "Hilbert_angle")
	{
		InitReconKernel_Hilbert << <(2 * N - 1 + 511) / 512, 512 >> > (reconKernel, N, du, kernelParam[0]);
	}
	else if (kernelName == "GaussianApodizedRamp")
	{
		InitReconKernel_GaussianApodized << <(2 * N - 1 + 511) / 512, 512 >> > (reconKernel, N, du, kernelParam[0]);
	}
	else if (kernelName == "None")
	{
		// Do not need to do anything
	}
}

void MallocManaged_Agent(float * &p, const int size)
{
	hipMallocManaged((void**)&p, size);
}


void CorrectBeamHardening_Agent(float* sgm, mango::Config & config)
{
	dim3 grid((config.sgmWidth + 15) / 16, (config.sgmHeight + 15) / 16);
	dim3 block(16, 16);

	CorrectBeamHardening_device << <grid, block >> > (sgm, config.sgmWidth, config.sgmHeight, config.sliceCount, config.beamHardening[0], config.beamHardening[1], config.beamHardening[2], config.beamHardening[3], config.beamHardening[4], config.beamHardening[5], config.beamHardening[6], config.beamHardening[7], config.beamHardening[8], config.beamHardening[9]);

	hipDeviceSynchronize();

}

void FilterSinogram_Agent(float * sgm, float* sgm_flt, float* reconKernel, float* u, mango::Config & config, float* beta, float * sdd_array, float * offcenter_array)
{
	// Step 1: weight the sinogram
	dim3 grid((config.sgmWidth + 15) / 16, (config.sgmHeight + 15) / 16);
	dim3 block(16, 16);

	// Hilbert kernel for phase contrast imaging
	if (config.kernelName == "Hilbert")
		WeightSinogramHilbert_device << <grid, block >> > (sgm, u, config.sgmWidth, config.sgmHeight, config.sliceCount, config.sdd);
	else if (config.kernelName == "Hilbert_angle")
	{
		printf("Kernel name: %s\n", config.kernelName);
		WeightSinogramHilbert_angle_device << <grid, block >> > (sgm, u, config.sgmWidth, config.sgmHeight, config.sliceCount, config.sdd);
	}
	else if (config.kernelName == "None")
	{
		// Do not weight the sinogram(sgm)
	}
	// Common attenuation imaging
	else
		WeightSinogram_device << <grid, block >> > (sgm, u, config.sgmWidth, config.sgmHeight, config.views, config.sliceCount, \
			config.sliceThickness, config.sliceOffcenter, sdd_array, config.totalScanAngle, config.shortScan, beta, offcenter_array);

	hipDeviceSynchronize();

	// Step 2: convolve the sinogram
	if (config.kernelName == "GaussianApodizedRamp")
	{
		// if Guassian aposied kernel is used, the sinogram need to be filtered twice
		// first by the ramp filter, then by the gaussian filter
		float du = config.detEltSize;
		float * reconKernel_ramp;
		hipMalloc((void**)&reconKernel_ramp, (2 * config.sgmWidth - 1) * sizeof(float));
		InitReconKernel_Hamming << <(2 * config.sgmWidth - 1 + 511) / 512, 512 >> > (reconKernel_ramp, config.sgmWidth, du, 1);

		hipDeviceSynchronize();

		//intermidiate filtration result is saved in sgm_flt_ramp
		float *sgm_flt_ramp;
		//MallocManaged_Agent(sgm_flt_ramp, config.sgmWidth*config.views*config.sliceCount * sizeof(float));
		hipMalloc((void**)& sgm_flt_ramp, config.sgmWidth * config.views * config.sliceCount * sizeof(float));

		ConvolveSinogram_device << <grid, block >> > (sgm_flt_ramp, sgm, reconKernel_ramp, config.sgmWidth, config.sgmHeight, config.views, config.sliceCount, u, config.detEltSize);
		hipDeviceSynchronize();
		//the height of the filtered sinogram shrinks to number of views, so the convolution parameters need to be adjusted accordingly
		ConvolveSinogram_device << <grid, block >> > (sgm_flt, sgm_flt_ramp, reconKernel, config.sgmWidth, config.views, config.views, config.sliceCount, u, config.detEltSize);
		hipDeviceSynchronize();

		// free temporary memory
		hipFree(reconKernel_ramp);
		hipFree(sgm_flt_ramp);
	}
	else if (config.kernelName == "None")
	{
		// Do not perfrom convolution, just directly copy the data
		CopySinogram_device <<<grid, block >>> (sgm_flt, sgm, config.sgmWidth, config.sgmHeight, config.views, config.sliceCount);
		hipDeviceSynchronize();
	}
	else
	{
		ConvolveSinogram_device << <grid, block >> > (sgm_flt, sgm, reconKernel, config.sgmWidth, config.sgmHeight, config.views, config.sliceCount, u, config.detEltSize);
		hipDeviceSynchronize();
	}
}

void BackprojectPixelDriven_Agent(float * sgm_flt, float * img, float* sdd_array, float* sid_array, float* offcenter_array, float* pmatrix_array, float * u, float *v, float* beta, mango::Config & config, int z_idx)
{
	dim3 grid((config.imgDim + 15) / 16, (config.imgDim + 15) / 16);
	dim3 block(16, 16);

	// Hilbert kernel for phase contrast imaging
	if (config.kernelName == "Hilbert" || config.kernelName == "Hilbert_angle")
	{
		BackprojectPixelDrivenHilbert_device << <grid, block >> > (sgm_flt, img, u, beta, config.sgmWidth, config.views, \
			config.sliceCount, config.imgDim, config.sdd, config.sid, config.detEltSize, config.pixelSize, config.xCenter, config.yCenter, z_idx);
	}
	// Common attenuation imaging
	else if (config.pmatrixFlag == false)// if pmatrix is not applied
	{
		BackprojectPixelDriven_device << <grid, block >> > (sgm_flt, img, u, v, beta, config.shortScan, config.sgmWidth, config.views, \
			config.sliceCount, config.coneBeam, config.imgDim, config.imgSliceCount, sdd_array, sid_array, offcenter_array, config.pixelSize, config.imgSliceThickness, \
			config.xCenter, config.yCenter, config.zCenter, z_idx);
	}
	else if (config.pmatrixFlag == true)// if pmatrix is applied
	{
		BackprojectPixelDriven_pmatrix_device << <grid, block >> > (sgm_flt, img, u, v, beta, pmatrix_array, config.shortScan, config.sgmWidth, config.views, \
			config.sliceCount, config.coneBeam, config.imgDim, config.imgSliceCount, sdd_array, sid_array, config.pixelSize, config.imgSliceThickness, \
			config.xCenter, config.yCenter, config.zCenter, z_idx, config.imgRot);
	}

	hipDeviceSynchronize();
}

void SaveReconImageSlice(const char* filename, float* rec_image, int z_idx, const mango::Config& config)
{
	FILE* fp = NULL;
	if (z_idx == 0)
		fp = fopen(filename, "wb");
	else
		fp = fopen(filename, "ab");

	if (fp == NULL)
	{
		fprintf(stderr, "Cannot save to file %s!\n", filename);
		exit(4);
	}
	fwrite(rec_image, sizeof(float), config.imgDim*config.imgDim, fp);
	fclose(fp);
}


void FreeMemory_Agent(float* &p)
{
	hipFree(p);
	p = nullptr;
}